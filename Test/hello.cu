#include "hip/hip_runtime.h"

#include <VectorAdd.cuh>

#include <stdio.h>
#include <thread>
#include <time.h>

void 
setGpu()
{
  //��ȡGPU�豸����
  int iDeviceCount = 0;
  hipGetDeviceCount(&iDeviceCount); 
  printf("GPU count: %d\n", iDeviceCount);
  
  //��ȡ����ʱ�豸��index 
  int iDev = 0;
  hipSetDevice(iDev);
  printf("GPU index: %d\n", iDev);
}

void 
initialData(float *arr, int elementCount)
{
  for(int i = 0; i < elementCount; i++)
  {
    arr[i] = i;
  }
}

void
vectorAddCpu(float *A, float *B, float *C, int N)
{
  for(int i = 0; i < N; i++)
  {
    C[i] = A[i] + B[i];
  }
}

int 
main()
{
  setGpu();

  unsigned long long iElemCount = 100000000;
  size_t stBytesCount = iElemCount*sizeof(float);
  float *fpHostA = (float *)malloc(stBytesCount);
  float *fpHostB = (float *)malloc(stBytesCount);
  float *fpHostC = (float *)malloc(stBytesCount);

  memset(fpHostA, 0, stBytesCount);
  memset(fpHostB, 0, stBytesCount);
  memset(fpHostC, 0, stBytesCount);

  float *fpDeviceA, *fpDeviceB, *fpDeviceC;
  hipMalloc(&fpDeviceA, stBytesCount);
  hipMalloc(&fpDeviceB, stBytesCount);
  hipMalloc(&fpDeviceC, stBytesCount);

  hipMemset(fpHostA, 0, stBytesCount);
  hipMemset(fpHostB, 0, stBytesCount);
  hipMemset(fpHostC, 0, stBytesCount);

  initialData(fpHostA, iElemCount);
  initialData(fpHostB, iElemCount);

  hipMemcpy(fpDeviceA, fpHostA, stBytesCount, hipMemcpyHostToDevice);
  hipMemcpy(fpDeviceB, fpHostB, stBytesCount, hipMemcpyHostToDevice);

  dim3 block(32);
  dim3 grid(iElemCount/32);

  clock_t startGpu, endGpu, startCpu, endCpu;

  startCpu = clock();
  vectorAddCpu(fpHostA, fpHostB, fpHostC, iElemCount);
  endCpu = clock();

  startGpu = clock();
  vectorAdd<<<grid, block>>>(fpDeviceA, fpDeviceB, fpDeviceC);
  endGpu = clock();
  hipDeviceSynchronize();

  printf("cpu:%f\ngpu:%f\n", (double)(endCpu - startCpu)/CLOCKS_PER_SEC, (double)(endGpu - startGpu)/CLOCKS_PER_SEC);
  
  hipMemcpy(fpHostA, fpDeviceA, stBytesCount, hipMemcpyDeviceToHost);
  hipMemcpy(fpHostB, fpDeviceB, stBytesCount, hipMemcpyDeviceToHost);
  hipMemcpy(fpHostC, fpDeviceC, stBytesCount, hipMemcpyDeviceToHost);

  // for(int i = 0; i < iElemCount; i++)
  // {
  //   printf("%f + %f = %f\n", fpHostA[i], fpHostB[i], fpHostC[i]);
  // }
} 